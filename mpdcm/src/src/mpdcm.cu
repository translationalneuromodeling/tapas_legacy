#include "hip/hip_runtime.h"
/* aponteeduardo@gmail.com */
/* copyright (C) 2014 */


#include "mpdcm.hcu"

// TODO move to the header

__device__
void 
dcm_upx_euler(dbuff ox, dbuff y, dbuff u, void *p_theta, void *p_ptheta, 
    dbuff nx);

__device__ 
void
dcm_upx_kr4(dbuff ox, dbuff y, dbuff u, void *p_theta, void *p_ptheta, 
    dbuff nx);

__device__
void 
dcm_upx_bs(dbuff ox, dbuff y, dbuff u, void *p_theta, void *p_ptheta, 
    dbuff nx, double* zs, thr_info tinfo);

__device__
void 
dcm_upx_bs0(dbuff ox, dbuff y, dbuff u, void *p_theta, void *p_ptheta, 
    dbuff nx);
__device__
void
bs_maxz(dbuff z);


// General functions

__device__ double dcm_dx(dbuff x, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, int i)
{
    double dx=0;
    double bt=0;
    int j;
    int k;
    int p;
    int o;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    o = INDEX_X * x.dim;

    // A
    for (j = 0; j < x.dim; j++)
    {
        dx += theta->A[i + x.dim*j] * x.arr[o + j];
    }

    for (j = 0; j < u.dim; j++)
    {
        if (  u.arr[j] == 0  )
            continue;
        // B
        bt = 0;
        k = x.dim*x.dim*j + i;
        for (p = 0; p < x.dim; p++){
            bt += theta->B[k + x.dim*p] * x.arr[o + p];
        }
        // C
        dx += (theta->C[i + x.dim*j] + bt)*u.arr[j];
    }

    return dx;
}

__device__ double dcm_ds(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int i)
{
    double ds;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    ds = x.arr[INDEX_X * x.dim + i] - 
        theta->K[i] * x.arr[INDEX_S * x.dim + i] -
        theta->gamma * (exp(x.arr[INDEX_F * x.dim + i]) - 1);

    return ds;
}

__device__ double dcm_df(dbuff x, dbuff y, dbuff u, void *p_theta, 
    void *p_ptheta, int i)
{
    double df;

    //ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    df = x.arr[INDEX_S * x.dim + i]*exp(-x.arr[INDEX_F * x.dim + i]);

    return df;
}

__device__ double dcm_dv(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int i)
{
    double dv;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    dv = exp(x.arr[INDEX_F * x.dim + i] - x.arr[INDEX_V * x.dim + i] - 
            theta->tau[i]) -
        exp(x.arr[INDEX_V * x.dim + i] * theta->alpha - theta->tau[i]);

    return dv;
}

__device__ double dcm_dq(dbuff x, dbuff y, dbuff u, void *p_theta, 
    void *p_ptheta, int i)
{
    double dq = 0;
    double f = exp(-x.arr[INDEX_F * x.dim + i]);
    double v;
    double lnE0; 
    ThetaDCM *theta = (ThetaDCM *) p_theta;
    
    v = exp(x.arr[INDEX_V * x.dim + i] * theta->alpha - theta->tau[i]);
    lnE0 = theta->lnE0;

    //    PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    dq = (1 - exp(f * theta->ln1_E0))*exp(x.arr[INDEX_F * x.dim + i] -
        lnE0 - theta->tau[i] -  x.arr[INDEX_Q * x.dim + i]) - v;

    return dq;
}

__device__ double dcm_lk1(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    double l;
    double q = exp(x.arr[INDEX_Q * x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k1 * ( 1 - q);

    return l;
}


__device__ double dcm_lk2(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    double l;
    double qv = exp(x.arr[INDEX_Q * x.dim + i] - x.arr[INDEX_V *x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k2 * ( 1 - qv);

    return l;
}

__device__ double dcm_lk3(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    double l;

    double v = exp(x.arr[INDEX_V * x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k3 * ( 1 - v);

    return l;
}

__device__ void dcm_upy(dbuff ox, dbuff y, dbuff u, void *theta,
    void *ptheta, dbuff nx)
{
    // Network node
    int j = threadIdx.x%y.dim;

    switch( threadIdx.y )
    {
        case INDEX_LK1 :
            nx.arr[ INDEX_LK1 * nx.dim + j] =
                dcm_lk1(ox, y, u, theta, ptheta, j);
            break;
        case INDEX_LK2:
            nx.arr[ INDEX_LK2 * nx.dim + j] =
                dcm_lk2(ox, y, u, theta, ptheta, j);
            break;
        case INDEX_LK3 :
            nx.arr[ INDEX_LK3 * nx.dim + j] =
                dcm_lk3(ox, y, u, theta, ptheta, j);
            break;
    }

}

// =======================================================================
// Integrators 
// =======================================================================

__device__ void dcm_int_euler(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int dp, unsigned int *errcode)
{
    int i;
    int j = threadIdx.x%y.dim;
    double *t;
    // Number of integration steps done between each data point
    int ss, dy;
    // Point where threads are not synchronized to anything
    int maxx = y.dim * (blockDim.x/y.dim);

    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;
    dbuff ox;
    dbuff nx;

    dbuff ty;
    dbuff tu;

    *errcode = 1;

    ox.dim = y.dim;
    nx.dim = y.dim;

    ox.arr = x.arr; 
    nx.arr = ox.arr + nx.dim * DIM_X;

    if ( threadIdx.x < maxx )
        memset(x.arr, 0, nx.dim * DIM_X * sizeof(double));

    __syncthreads();
    ty.dim = y.dim;
    tu.dim = u.dim;

    // How many samples are gonna be taken
    ss = ceil(1.0/ptheta->dt);
    dy = ceil(1.0/(ptheta->dt*ptheta->dyu));

    ty.arr = y.arr; 
    tu.arr = u.arr;

    for (i=0; i < dp*ss; i++)
    {
        if ( threadIdx.x < maxx )
            dcm_upx_euler(ox, ty, tu, p_theta, p_ptheta, nx);
        __syncthreads();
        // Only sample every 1/ptheta->dt times
        if ( i%ss == 0 )
        {
           if ( i%dy == 0 ) 
           {
                if ( threadIdx.x < maxx )
                    dcm_upy(nx, ty, tu, p_theta, p_ptheta, ox);           
                __syncthreads();
                if ( threadIdx.x < maxx && threadIdx.y == 0 )
                    ty.arr[j] = ox.arr[INDEX_LK1 * ox.dim + j] +
                        ox.arr[ INDEX_LK2 * ox.dim + j] +
                        ox.arr[ INDEX_LK3 * ox.dim + j];
                __syncthreads();

                ty.arr += y.dim; 
            }
            if ( i > 0 )
                tu.arr += u.dim;
        }
        // Swap the pointers
        t = ox.arr;
        ox.arr = nx.arr;
        nx.arr = t;
    }

    *errcode = 0;
}

// Runge Kutta

__device__ void dcm_int_kr4(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int dp, unsigned int *errcode)
{
    int i;
    int j = threadIdx.x%y.dim;
    double *t;
    // Number of integration steps done between each data point
    int ss, dy;
    // Point where threads are not synchronized to anything
    int maxx = y.dim * (blockDim.x/y.dim);

    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;
    dbuff ox;
    dbuff nx;

    dbuff ty;
    dbuff tu;

    *errcode = 0;

    ox.dim = y.dim;
    nx.dim = y.dim;

    ox.arr = x.arr; 
    nx.arr = ox.arr + nx.dim * DIM_X;

    if ( threadIdx.x < maxx )
        memset(x.arr, 0, nx.dim * DIM_X * sizeof(double));

    __syncthreads();
    ty.dim = y.dim;
    tu.dim = u.dim;

    // How many samples are gonna be taken
    ss = ceil(1.0/ptheta->dt);
    dy = ceil(1.0/(ptheta->dt*ptheta->dyu));

    ty.arr = y.arr; 
    tu.arr = u.arr;

    for (i=0; i < dp*ss; i++)
    {
        //if ( threadIdx.x < maxx )
            dcm_upx_kr4(ox, ty, tu, p_theta, p_ptheta, nx);
        __syncthreads();
        // Only sample every 1/ptheta->dt times
        if ( i%ss == 0 )
        {
            if ( i%dy == 0 ) 
            {
                if ( threadIdx.x < maxx )
                    dcm_upy(nx, ty, tu, p_theta, p_ptheta, ox);           
                __syncthreads();
                if ( threadIdx.x < maxx && threadIdx.y == 0 )
                    ty.arr[j] = ox.arr[INDEX_LK1 * ox.dim + j] +
                        ox.arr[ INDEX_LK2 * ox.dim + j] +
                        ox.arr[ INDEX_LK3 * ox.dim + j];
                if ( i > 0 )
                    ty.arr += y.dim;
               __syncthreads(); 
            }
            if ( i > 0 )
                tu.arr += u.dim;
        }
        // Swap the pointers
        t = ox.arr;
        ox.arr = nx.arr;
        nx.arr = t;
    }
    *errcode = 0;
}

#define MINDY 1
#define MAXDY 64

#define MINTOL 0.000001
#define MAXTOL 0.001000


// Bucacki Shampinee

__device__ void dcm_int_bs(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int dp, unsigned int *errcode, thr_info tinfo)
{
    int i;
    int j = threadIdx.x%y.dim;
    double *t;
    double z;
    // Number of integration steps done between each output
    int dy;
    // Point where threads are not synchronized to anything
    int maxx = threadIdx.x - y.dim * (blockDim.x/y.dim);
    unsigned int ndt = MAXDY, odt = MAXDY;

    __shared__ double zs[NUM_THREADS];

    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;
    dbuff ox;
    dbuff nx;

    dbuff ty;
    dbuff tu;

    *errcode = 0;

    ox.dim = y.dim;
    nx.dim = y.dim;

    ox.arr = x.arr; 
    nx.arr = ox.arr + nx.dim * DIM_X;

    if ( maxx < 0 )
        memset(x.arr, 0, nx.dim * DIM_X * sizeof(double));

    // Restart the errors
    if ( threadIdx.y == 0 )
        zs[threadIdx.x] = 0;

    __syncthreads();

    ty.dim = y.dim;
    tu.dim = u.dim;
    ty.arr = y.arr; 
    tu.arr = u.arr;

    // How many samples are gonna be taken

    dy = MAXDY * ceil(1 / ptheta->dyu);
    ptheta->de = 2*ptheta->dyu;
    __syncthreads();

    dcm_upx_bs0(ox, ty, tu, p_theta, p_ptheta, nx);
    __syncthreads();

    i = 0;

    while ( i < dp * MAXDY + 1 )
    {
        dcm_upx_bs(ox, ty, tu, p_theta, p_ptheta, nx, zs, tinfo);

        __syncthreads();

        z = zs[0];

        __syncthreads();

        // Exceeded the error tolerance
        if ( z > MAXTOL && odt > MINDY )
        {
            odt >>= 1;
            if ( threadIdx.x == 0 && threadIdx.y == 0 )
                ptheta->de = 2 * ptheta->dyu * (((float ) odt)/MAXDY);   
        
            __syncthreads();
        
            continue;
        }
        
        // Below the error tolerance

        if ( z < MINTOL && odt < MAXDY )
        {
            odt <<= 1;
        }

        // Always sample at the right spot.
        if ( i%MAXDY + odt > MAXDY )
            ndt = MAXDY - i%MAXDY;
        else 
            ndt = odt;

        if ( threadIdx.x == 0 && threadIdx.y == 0 )
            ptheta->de = 2*ptheta->dyu * (((float ) ndt)/MAXDY);
       
        __syncthreads();

        // Only sample every 1/ptheta->dt times
        if ( i%MAXDY == 0 )
        {
            if ( i%dy == 0 ) 
            {
               if ( maxx < 0 )
                    dcm_upy(nx, ty, tu, p_theta, p_ptheta, ox);           
                __syncthreads();
               if ( maxx < 0 && threadIdx.y == 0 )
                   ty.arr[j] = ox.arr[INDEX_LK1 * ox.dim + j] +
                       ox.arr[ INDEX_LK2 * ox.dim + j] +
                       ox.arr[ INDEX_LK3 * ox.dim + j];
                if ( i > 0 )
                    ty.arr += y.dim;
               __syncthreads(); 
            }
            if ( i > 0 )
                tu.arr += u.dim;
        }
        // Swap the pointers
        t = ox.arr;
        ox.arr = nx.arr;
        nx.arr = t;

        i += ndt;
    }
    *errcode = 0;

}


// ==========================================================================
// Kernel code
// ==========================================================================

__global__ void kdcm_euler(double *x, double *y, double *u, 
    void *p_theta, double *d_theta, void *p_ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb, unsigned int *errcode)
{

    int i;
    dbuff tx, ty, tu;
    extern __shared__ double sx[];

    // Assign pointers to theta


    ThetaDCM *theta = (ThetaDCM *) p_theta;
    ThetaDCM *ltheta;

    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;
    __shared__ PThetaDCM lptheta[1];

    lptheta->dt = ptheta->dt;
    lptheta->dyu = ptheta->dyu;
    lptheta->de = ptheta->de;
    lptheta->mode = ptheta->mode;

    tu.dim = nu;
    tx.dim = nx; 
    ty.dim = nx;

    i = threadIdx.x/nx + (blockDim.x / nx) * blockIdx.x;
    // Iterate in chuncks
    while ( i < nb * nt )
    {
        double *o;

        tu.arr = u + (i/nb) * nu * dp;
        // Get the new address

        ltheta = theta + i;

        o = d_theta + i * (
            nx * nx + // A
            nx * nx * nu + // B 
            nx * nu + // C
            nx + // Kappa (K)
            nx); // tau
        
        ltheta->A = o;
        o += nx * nx;

        ltheta->B = o;
        o += nx * nx * nu;

        ltheta->C = o; 
        o+= nx * nu;

        ltheta->K = o;
        o += nx;

        ltheta->tau = o; 

        tx.arr = sx + PRELOC_SIZE_X_EULER * DIM_X * nx * (threadIdx.x/nx);

        ty.arr = y + i * nx * ny;
        dcm_int_euler(tx, ty, tu, (void *) ltheta, (void *) lptheta, dp, 
            errcode);

        i += gridDim.x * (blockDim.x / nx );        
    }
}

__global__ void kdcm_kr4(double *x, double *y, double *u, 
    void *p_theta, double *d_theta, void *p_ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb, unsigned int *errcode)
{

    int i;
    dbuff tx, ty, tu;
    extern __shared__ double sx[];

    // Assign pointers to theta


    ThetaDCM *theta = (ThetaDCM *) p_theta;
    ThetaDCM *ltheta;

    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;
    __shared__ PThetaDCM lptheta[1];

    lptheta->dt = ptheta->dt;
    lptheta->dyu = ptheta->dyu;
    lptheta->de = ptheta->de;
    lptheta->mode = ptheta->mode;

    tu.dim = nu;
    tx.dim = nx; 
    ty.dim = nx;

    i = threadIdx.x/nx + (blockDim.x / nx) * blockIdx.x;
    // Iterate in chuncks
    while ( i < nb * nt )
    {
        double *o;

        tu.arr = u + (i/nb) * nu * dp;
        // Get the new address

        ltheta = theta + i;

        o = d_theta + i * (
            nx * nx + // A
            nx * nx * nu + // B 
            nx * nu + // C
            nx + // Kappa (K)
            nx); // tau
        
        ltheta->A = o;
        o += nx * nx;

        ltheta->B = o;
        o += nx * nx * nu;

        ltheta->C = o; 
        o+= nx * nu;

        ltheta->K = o;
        o += nx;

        ltheta->tau = o; 

        tx.arr = sx + PRELOC_SIZE_X_KR4 * DIM_X * nx * (threadIdx.x/nx);

        ty.arr = y + i * nx * ny;
        dcm_int_kr4(tx, ty, tu, (void *) ltheta, (void *) lptheta, dp, errcode);
        i += gridDim.x * (blockDim.x / nx );        
    }
}

__global__ void kdcm_bs(double *x, double *y, double *u, 
    void *p_theta, double *d_theta, void *p_ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb, unsigned int * errcode)
{
    /* 
    mem -- Prealocate shared memory. It depends on the slots that the 
        integrator needs; two for euler and 4 for Kutta-Ruge.
    fupx -- Function used to integrate the update the system. 
    */

    int i;
    dbuff tx, ty, tu;
    extern __shared__ double sx[];

    // Assign pointers to theta

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    ThetaDCM *ltheta;

    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;
    __shared__ PThetaDCM lptheta[1];

    thr_info tinfo[1];

    tinfo->ns = nt * nb;

    lptheta->dt = ptheta->dt;
    lptheta->dyu = ptheta->dyu;
    lptheta->de = ptheta->de;
    lptheta->mode = ptheta->mode;

    tu.dim = nu;
    tx.dim = nx; 
    ty.dim = nx;

    i = threadIdx.x/nx + (blockDim.x / nx) * blockIdx.x;
    // Iterate in chuncks
    while ( i < nb * nt )
    {
        double *o;

        tinfo->cs = i;

        tu.arr = u + (i/nb) * nu * dp;
        // Get the new address

        ltheta = theta + i;

        o = d_theta + i * (
            nx * nx + // A
            nx * nx * nu + // B 
            nx * nu + // C
            nx + // Kappa (K)
            nx); // tau
        
        ltheta->A = o;
        o += nx * nx;

        ltheta->B = o;
        o += nx * nx * nu;

        ltheta->C = o; 
        o+= nx * nu;

        ltheta->K = o;
        o += nx;

        ltheta->tau = o; 

        tx.arr = sx + PRELOC_SIZE_X_BS * DIM_X * nx * (threadIdx.x/nx);

        ty.arr = y + i * nx * ny;
        dcm_int_bs(tx, ty, tu, (void *) ltheta, (void *) lptheta, dp, 
            errcode, *tinfo);
        i += gridDim.x * (blockDim.x / nx );        
    }
}


// ===========================================================================
// Kernel (l)auncher
// ===========================================================================


__host__ void ldcm_euler
(double *x, double *y, double *u, 
    void *theta, double *d_theta, void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb, unsigned int *errcode)
{

    int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    dim3 gthreads(NUM_THREADS, DIM_X);
    dim3 gblocks(NUM_BLOCKS * props.multiProcessorCount, 1);

    int sems;
    sems =  NUM_THREADS * DIM_X * PRELOC_SIZE_X_EULER * sizeof( double );


    kdcm_euler<<<gblocks, gthreads, sems>>>(x, y, u, 
        theta, d_theta, ptheta, d_ptheta, 
        nx, ny, nu, dp, nt, nb, errcode); 
}


__host__ void
ldcm_kr4(double *x, double *y, double *u, 
    void *theta, double *d_theta, void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb, unsigned int *errcode)
{
    int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    dim3 gthreads(NUM_THREADS, DIM_X);
    dim3 gblocks(NUM_BLOCKS * props.multiProcessorCount, 1);

    int smems = NUM_THREADS * DIM_X * PRELOC_SIZE_X_KR4 * sizeof( double );

    kdcm_kr4<<<gblocks, gthreads, smems>>>(x, y, u, 
        theta, d_theta, ptheta, d_ptheta, 
        nx, ny, nu, dp, nt, nb, errcode); 
}

__host__ void 
ldcm_bs(double *x, double *y, double *u, 
    void *theta, double *d_theta, void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb, unsigned int *errcode)
{

    int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    dim3 gthreads(NUM_THREADS, DIM_X);
    dim3 gblocks(NUM_BLOCKS * props.multiProcessorCount, 1);

    int smems = NUM_THREADS * DIM_X * PRELOC_SIZE_X_BS * sizeof( double );

    kdcm_bs<<<gblocks, gthreads, smems>>>(x, y, u, 
        theta, d_theta, ptheta, d_ptheta, 
        nx, ny, nu, dp, nt, nb, errcode); 
}


// ===========================================================================
// Allocate memory
// ===========================================================================

// Device alloc memory theta
__host__ 
void
dam_theta(
    void **theta, double **d_theta,
    void **pd_theta, double **dd_theta,
    int nx, int ny, int nu, int dp, int nt, int nb)
{

    int tp;

    // Allocate memory for the structures

    HANDLE_ERROR( hipMalloc( pd_theta, nt * nb * sizeof(ThetaDCM)));
    HANDLE_ERROR( hipMemcpy( *pd_theta, *theta, nt * nb * sizeof(ThetaDCM),
        hipMemcpyHostToDevice ) );

    // Allocate memory for the matrices. It is assumed that the parameters
    // are in a block of contiguous memory with the for A, Bs, C, x0, epsilon,
    // Kappa, tau

    tp = nt * nb * 
        (nx * nx +      // A:
        nx * nx * nu +  // B's
        nx * nu +       // C
        nx + // kappa
        nx); // tau 

    HANDLE_ERROR( hipMalloc( dd_theta, tp * sizeof(double) ) );
    HANDLE_ERROR( hipMemcpy( (void *) *dd_theta, (void *) *d_theta, 
        tp * sizeof(double), hipMemcpyHostToDevice ) );

}

// Device alloc memory ptheta
__host__ 
void 
dam_ptheta(
    void **ptheta, double **d_ptheta, 
    void **pd_ptheta, double **dd_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb)
{

    HANDLE_ERROR( hipMalloc( pd_ptheta, sizeof(PThetaDCM)));
    HANDLE_ERROR( hipMemcpy( *pd_ptheta, *ptheta, sizeof(PThetaDCM),
        hipMemcpyHostToDevice ) );
 
}

// ===========================================================================
// Host code
// ===========================================================================

extern "C"
int 
mpdcm_fmri( double *x, double *y, double *u,
    void *theta, double *d_theta,
    void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb,
    klauncher launcher)
{

    double *d_x, *d_y, *d_u;
    void *pd_theta, *pd_ptheta;
    double *dd_theta, *dd_ptheta;
    unsigned int errcode[1], *d_errcode;

    // x

    d_x = 0;

    // y

    HANDLE_ERROR( hipMalloc( (void **) &d_y,
        nx * ny * nt * nb * sizeof(double) ) );

    // u

    HANDLE_ERROR( hipMalloc( (void**) &d_u,
        nt * nu * dp *  sizeof(double) ) );

    HANDLE_ERROR( hipMemcpy( d_u, u, nt * nu * dp * sizeof(double),
        hipMemcpyHostToDevice ) );

    // Error code

    HANDLE_ERROR( hipMalloc( (void**) &d_errcode, 
        sizeof( unsigned int ) ) );


    // Theta 
    dam_theta(
        &theta, &d_theta,
        &pd_theta, &dd_theta,
        nx, ny, nu, dp, nt, nb);
    
    // PThetaDCM

    dam_ptheta(
        &ptheta, &d_ptheta,
        &pd_ptheta, &dd_ptheta,
        nx, ny, nu, dp, nt, nb); 

    // Launch the kernel
    (*launcher)(
        d_x, d_y, d_u, 
        pd_theta, dd_theta, 
        pd_ptheta, dd_ptheta,
        nx, ny, nu, dp, nt, nb, d_errcode);

    // Get y back

    HANDLE_ERROR( hipMemcpy(y, d_y,
        nx * ny * nt * nb * sizeof(double),
        hipMemcpyDeviceToHost) );

    HANDLE_ERROR( hipMemcpy(errcode, d_errcode, sizeof( unsigned int ), 
        hipMemcpyDeviceToHost) );

    if ( *errcode != 0 ) 
        printf( "Error %d in %s at line %d\n", *errcode, __FILE__, __LINE__ );

    // free the memory allocated on the GPU
    //HANDLE_ERROR( hipFree( d_x ) );
    HANDLE_ERROR( hipFree( d_y ) );
    HANDLE_ERROR( hipFree( d_u ) );
    HANDLE_ERROR( hipFree( d_errcode ) );

    HANDLE_ERROR( hipFree( pd_theta ) );
    HANDLE_ERROR( hipFree( dd_theta ) );

    if ( DIM_PTHETA ) HANDLE_ERROR( hipFree( pd_ptheta ) );
    if ( DIM_DPTHETA ) HANDLE_ERROR( hipFree( dd_ptheta ) );
    
    return 0; 
}

// =======================================================================
// Externals
// =======================================================================

extern "C"
int
mpdcm_fmri_euler( double *x, double *y, double *u,
    void *theta, double *d_theta,
    void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb)
{
   int r = mpdcm_fmri(x, y, u,
        theta, d_theta,
        ptheta, d_ptheta, 
        nx, ny, nu, dp, nt, nb,
        &ldcm_euler);
    
    return r;
};

extern "C"
int
mpdcm_fmri_kr4( double *x, double *y, double *u,
    void *theta, double *d_theta,
    void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb)
{
    int r = mpdcm_fmri(x, y, u,
        theta, d_theta,
        ptheta, d_ptheta, 
        nx, ny, nu, dp, nt, nb,
        &ldcm_kr4);

    return r;
}

extern "C"
int
mpdcm_fmri_bs( double *x, double *y, double *u,
    void *theta, double *d_theta,
    void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb)
{
    int r = mpdcm_fmri(x, y, u,
        theta, d_theta,
        ptheta, d_ptheta, 
        nx, ny, nu, dp, nt, nb,
        &ldcm_bs);

    return r;
};

// =======================================================================
// Integrators
// =======================================================================

// The memory layout is the following:

// If it is a 4 region DCM if work the following way:
// x_1, x_2, x_3, x_4, f_1, f_2, f_3, f_4, ....


// Euler

__device__ void dcm_upx_euler(dbuff ox, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, dbuff nx)
{

    //ThetaDCM *theta = (ThetaDCM *) p_theta;
    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    int j = threadIdx.x%y.dim;
    int s;
    int maxx = threadIdx.x - y.dim * (blockDim.x/y.dim);


    // Make the values to be closer in range
 
    if ( isnan( *u.arr ) ){
        if ( threadIdx.y == 0 && maxx < 0 )
        {
            nx.arr[ INDEX_X * ox.dim + j] = NAN;
            nx.arr[ INDEX_F * ox.dim + j] = NAN;
            nx.arr[ INDEX_S * ox.dim + j] = NAN;
            nx.arr[ INDEX_V * ox.dim + j] = NAN;
            nx.arr[ INDEX_Q * ox.dim + j] = NAN;
        }
    }
    if ( maxx < 0 )
    {
        switch ( threadIdx.y )
        {
            case INDEX_X:
                s = INDEX_X * ox.dim + j;
                nx.arr[s] = ox.arr[s] + 
                    ptheta->de * dcm_dx(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_F:
                s = INDEX_F * ox.dim + j;
                nx.arr[s] = ox.arr[s] + 
                    ptheta->de * dcm_df(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_S:
                s = INDEX_S * ox.dim + j;
                nx.arr[s] = ox.arr[s] + 
                    ptheta->de * dcm_ds(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_V:
                s = INDEX_V * ox.dim + j;
                nx.arr[s] = ox.arr[s] + 
                    ptheta->de * dcm_dv(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_Q:
                s = INDEX_Q * ox.dim + j;
                nx.arr[s] = ox.arr[s] + 
                    ptheta->de * dcm_dq(ox, y, u, p_theta, p_ptheta, j); 
                break;
        }
    }
}

// Runge Kutta

__device__ void dcm_upx_kr4(dbuff ox, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, dbuff nx)
{

    //ThetaDCM *theta = (ThetaDCM *) p_theta;
    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    int j = threadIdx.x%y.dim;
    int s;
    int maxx = threadIdx.x - y.dim * (blockDim.x / y.dim);
    // Buffers for the intermediate results. z is the estimated error.
    dbuff k1, k2;

    k1.arr = (ox.arr < nx.arr) ? ox.arr : nx.arr;
    k2.arr = k1.arr;
    k1.arr += ox.dim * DIM_X * 2;  
    k2.arr += ox.dim * DIM_X * 3;

    k1.dim = ox.dim;
    k2.dim = ox.dim;

    // Make the values to be closer in range
 
    if ( isnan( *u.arr ) ){
        if ( threadIdx.y == 0 && maxx < 0 )
        {
            nx.arr[ INDEX_X * ox.dim + j] = NAN;
            nx.arr[ INDEX_F * ox.dim + j] = NAN;
            nx.arr[ INDEX_S * ox.dim + j] = NAN;
            nx.arr[ INDEX_V * ox.dim + j] = NAN;
            nx.arr[ INDEX_Q * ox.dim + j] = NAN;
        }
    }
    if ( maxx < 0 )
    {
        switch ( threadIdx.y )
        {
            case INDEX_X:
                s = INDEX_X * ox.dim + j;
                k1.arr[s] = dcm_dx(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_F:
                s = INDEX_F * ox.dim + j;
                k1.arr[s] = dcm_df(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_S:
                s = INDEX_S * ox.dim + j;
                k1.arr[s] = dcm_ds(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_V:
                s = INDEX_V * ox.dim + j;
                k1.arr[s] = dcm_dv(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_Q:
                s = INDEX_Q * ox.dim + j;
                k1.arr[s] = dcm_dq(ox, y, u, p_theta, p_ptheta, j); 
                break;
        } 
        nx.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s] * KRW1;
        k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
    }
    __syncthreads();

    if ( maxx < 0 )
    {
        switch ( threadIdx.y )
        {
            case INDEX_X:
                k1.arr[s] = dcm_dx(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_F:
                k1.arr[s] = dcm_df(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_S:
                k1.arr[s] = dcm_ds(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_V:
                k1.arr[s] = dcm_dv(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_Q:
                k1.arr[s] = dcm_dq(k2, y, u, p_theta, p_ptheta, j); 
                break;
        }

        nx.arr[s] += ptheta->de * k1.arr[s] * KRW2;
    }

    __syncthreads();

    if ( maxx < 0 )
        k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 

    __syncthreads();

    if ( maxx < 0 )
    {
        switch ( threadIdx.y )
        {
            case INDEX_X:
                k1.arr[s] = dcm_dx(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_F:
                k1.arr[s] = dcm_df(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_S:
                k1.arr[s] = dcm_ds(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_V:
                k1.arr[s] = dcm_dv(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_Q:
                k1.arr[s] = dcm_dq(k2, y, u, p_theta, p_ptheta, j); 
                break;
        }

        nx.arr[s] += ptheta->de * k1.arr[s]*KRW3;

    }

    __syncthreads();

    if ( maxx < 0 )
        k2.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s];

    __syncthreads();
    
    if ( maxx < 0 )
    {
        switch ( threadIdx.y )
        {
            case INDEX_X:
                k1.arr[s] = dcm_dx(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_F:
                k1.arr[s] = dcm_df(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_S:
                k1.arr[s] = dcm_ds(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_V:
                k1.arr[s] = dcm_dv(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_Q:
                k1.arr[s] = dcm_dq(k2, y, u, p_theta, p_ptheta, j); 
                break;
        }
        nx.arr[s] += ptheta->de * k1.arr[s] * KRW4;
    }
}


// Bogacki Shampine

__device__ void dcm_upx_bs(dbuff ox, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, dbuff nx, double *zs, thr_info tinfo)
{

    //ThetaDCM *theta = (ThetaDCM *) p_theta;
    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    int j = threadIdx.x%y.dim;
    int s;
    int maxx = threadIdx.x - y.dim * ( blockDim.x / y.dim );
    // Buffers for the intermediate results. z is the estimated error.
    dbuff k1, k2, z;

    k1.arr = (ox.arr < nx.arr) ? ox.arr : nx.arr;
    k2.arr = k1.arr;
    z.arr = k1.arr;

    k1.arr += ox.dim * DIM_X * 2;  
    k2.arr += ox.dim * DIM_X * 3;
    z.arr += ox.dim * DIM_X * 4;

    k1.dim = ox.dim;
    k2.dim = ox.dim;
    z.dim = ox.dim;

    // Make the values to be closer in range
 
    if ( isnan( *u.arr ) ){
        if ( threadIdx.y == 0  && maxx < 0 )
        {
            nx.arr[ INDEX_X * ox.dim + j] = NAN;
            nx.arr[ INDEX_F * ox.dim + j] = NAN;
            nx.arr[ INDEX_S * ox.dim + j] = NAN;
            nx.arr[ INDEX_V * ox.dim + j] = NAN;
            nx.arr[ INDEX_Q * ox.dim + j] = NAN;
        }
    }

    // Memory 
    if ( maxx < 0 )
    {
        switch ( threadIdx.y )
        {
            case INDEX_X:
                s = INDEX_X * ox.dim + j;
                break;
            case INDEX_F:
                s = INDEX_F * ox.dim + j;
                break;
            case INDEX_S:
                s = INDEX_S * ox.dim + j;
                break;
            case INDEX_V:
                s = INDEX_V * ox.dim + j;
                break;
            case INDEX_Q:
                s = INDEX_Q * ox.dim + j;
                break;
        }

        k2.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s] * BSA1;
        nx.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s] * BSB1;
        z.arr[s] = k1.arr[s] * BSZ1;
    }

    __syncthreads();
    
    if ( maxx < 0 )
    {
        switch ( threadIdx.y )
        {
            case INDEX_X:
                k1.arr[s] = dcm_dx(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_F:
                k1.arr[s] = dcm_df(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_S:
                k1.arr[s] = dcm_ds(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_V:
                k1.arr[s] = dcm_dv(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_Q:
                k1.arr[s] = dcm_dq(k2, y, u, p_theta, p_ptheta, j); 
                break;
        }
        nx.arr[s] += ptheta->de * k1.arr[s] * BSB2;
        z.arr[s] += k1.arr[s] * BSZ2;
    }
    
    __syncthreads();

    // Synchronize memory
    if ( maxx < 0 )
        k2.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s] * BSA2; 

    __syncthreads();

    if ( maxx < 0 )
    {
        switch ( threadIdx.y )
        {
            case INDEX_X:
                k1.arr[s] = dcm_dx(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_F:
                k1.arr[s] = dcm_df(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_S:
                k1.arr[s] = dcm_ds(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_V:
                k1.arr[s] = dcm_dv(k2, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_Q:
                k1.arr[s] = dcm_dq(k2, y, u, p_theta, p_ptheta, j); 
                break;
        }

        nx.arr[s] += ptheta->de * k1.arr[s] * BSB3;
        z.arr[s] += k1.arr[s] * BSZ3;
    }
    
    __syncthreads();

    if ( maxx < 0 )
    {
        switch ( threadIdx.y )
        {
            case INDEX_X:
                k1.arr[s] = dcm_dx(nx, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_F:
                k1.arr[s] = dcm_df(nx, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_S:
                k1.arr[s] = dcm_ds(nx, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_V:
                k1.arr[s] = dcm_dv(nx, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_Q:
                k1.arr[s] = dcm_dq(nx, y, u, p_theta, p_ptheta, j); 
                break;
        }
        z.arr[s] += k1.arr[s] * BSZ4;
        z.arr[s] *= ptheta->de;
        z.arr[s] = abs(z.arr[s]);
    }

    __syncthreads();

    if ( maxx < 0 && threadIdx.y == 0) 
        zs[threadIdx.x] = z.arr[s];
//    if ( tinfo.cs >= tinfo.ns && threadIdx.y == 0 )
//        zs[threadIdx.x] = 0;
    // This is a serious hack
    if ( threadIdx.x >= z.dim && threadIdx.y == 0 )
        zs[threadIdx.x] = 0;

    __syncthreads();
    if ( threadIdx.x < 16 ) 
        zs[threadIdx.x] = zs[threadIdx.x] > zs[threadIdx.x + 16] ? 
            zs[threadIdx.x] : zs[threadIdx.x + 16];
    __syncthreads();
    if ( threadIdx.x < 8 ) 
        zs[threadIdx.x] = zs[threadIdx.x] > zs[threadIdx.x + 8] ? 
            zs[threadIdx.x] : zs[threadIdx.x + 8];
    __syncthreads();
   if ( threadIdx.x < 4 ) 
        zs[threadIdx.x] = zs[threadIdx.x] > zs[threadIdx.x + 4] ? 
            zs[threadIdx.x] : zs[threadIdx.x + 4];
    __syncthreads();
    if ( threadIdx.x < 2 ) 
        zs[threadIdx.x] = zs[threadIdx.x] > zs[threadIdx.x + 2] ? 
            zs[threadIdx.x] : zs[threadIdx.x + 2];
    __syncthreads();
    if ( threadIdx.x == 0 ) 
        zs[threadIdx.x] = zs[threadIdx.x] > zs[threadIdx.x + 1] ? 
            zs[threadIdx.x] : zs[threadIdx.x + 1];
    __syncthreads();
}


__device__ void dcm_upx_bs0(dbuff ox, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, dbuff nx)
{
    // Compute the value of f for the first iteration. This is neede only to
    // initilize the integrator.

    //ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    int j = threadIdx.x%y.dim;
    int s;
    
    // Buffers for the intermediate results.

    dbuff k1;

    int maxx = threadIdx.x - y.dim * (blockDim.x/y.dim);

    k1.arr = (ox.arr < nx.arr) ? ox.arr : nx.arr;
    k1.arr += ox.dim * DIM_X * 2;  
    k1.dim = ox.dim;

    // Make the values to be closer in range
 
    if ( isnan( *u.arr ) ){
        if ( threadIdx.y == 0  && maxx < 0 )
        {
            nx.arr[ INDEX_X * ox.dim + j] = NAN;
            nx.arr[ INDEX_F * ox.dim + j] = NAN;
            nx.arr[ INDEX_S * ox.dim + j] = NAN;
            nx.arr[ INDEX_V * ox.dim + j] = NAN;
            nx.arr[ INDEX_Q * ox.dim + j] = NAN;
        }
    }

    if ( maxx < 0 )
    {
        switch ( threadIdx.y )
        {
            case INDEX_X:
                s = INDEX_X * ox.dim + j;
                k1.arr[s] = dcm_dx(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_F:
                s = INDEX_F * ox.dim + j;
                k1.arr[s] = dcm_df(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_S:
                s = INDEX_S * ox.dim + j;
                k1.arr[s] = dcm_ds(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_V:
                s = INDEX_V * ox.dim + j;
                k1.arr[s] = dcm_dv(ox, y, u, p_theta, p_ptheta, j);
                break;
            case INDEX_Q:
                s = INDEX_Q * ox.dim + j;
                k1.arr[s] = dcm_dq(ox, y, u, p_theta, p_ptheta, j); 
                break;
        }
    }
}

