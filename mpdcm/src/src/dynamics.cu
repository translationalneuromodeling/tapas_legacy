#include "hip/hip_runtime.h"
//
// Author: Eduardo Aponte
// Copyright 2015 by Eduardo Aponte <aponteeduardo@gmail.com>
//
// Licensed under GNU General Public License 3.0 or later.
// Some rights reserved. See COPYING, AUTHORS.
//
// Revision log:
//


#include "mpdcm.hcu"

// General functions

__device__
MPFLOAT
dcm_dx(dbuff x, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, int i)
{
    MPFLOAT dx = 0;
    MPFLOAT bt = 0;
    int j;
    int k;
    int p;
    int o;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    o = INDEX_X * x.dim;

    // A
    for (j = 0; j < x.dim; j++)
    {
        dx = fma(theta->A[i + x.dim*j], x.arr[o + j], dx);
        if ( theta->fD == MF_TRUE )
        {
            bt = 0;
            k = x.dim * x.dim * j + i;
            for (p = 0; p < x.dim; p++)
                bt = fma(theta->D[k + x.dim * p], x.arr[o + p], bt);
            dx = fma(bt, x.arr[o + j], dx);
        }
    }

    for (j = 0; j < u.dim; j++)
    {
        if (  u.arr[j] == 0  )
            continue;
        // B
        bt = 0;
        k = x.dim * x.dim * j + i;
        for (p = 0; p < x.dim; p++){
            bt = fma(theta->B[k + x.dim * p], x.arr[o + p], bt);
        }
        // C
        dx = fma(theta->C[i + x.dim * j] + bt, u.arr[j], dx);
    }

    return dx;
}

__device__ 
MPFLOAT 
dcm_ds(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int i)
{
    MPFLOAT ds;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    ds = x.arr[INDEX_X * x.dim + i] - 
        theta->K[i] * x.arr[INDEX_S * x.dim + i] -
        theta->gamma * (exp(x.arr[INDEX_F * x.dim + i]) - 1);

    return ds;
}

__device__
MPFLOAT
dcm_df(dbuff x, dbuff y, dbuff u, void *p_theta, 
    void *p_ptheta, int i)
{
    MPFLOAT df;

    //ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    df = x.arr[INDEX_S * x.dim + i]*exp(-x.arr[INDEX_F * x.dim + i]);

    return df;
}

__device__
MPFLOAT 
dcm_dv(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int i)
{
    MPFLOAT dv;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    dv = exp(x.arr[INDEX_F * x.dim + i] - x.arr[INDEX_V * x.dim + i] - 
            theta->tau[i]) -
        exp(x.arr[INDEX_V * x.dim + i] * theta->alpha - theta->tau[i]);

    return dv;
}

__device__
MPFLOAT
dcm_dq(dbuff x, dbuff y, dbuff u, void *p_theta, 
    void *p_ptheta, int i)
{
    MPFLOAT dq = 0;
    MPFLOAT f = exp(-x.arr[INDEX_F * x.dim + i]);
    MPFLOAT v;
    MPFLOAT lnE0; 
    ThetaDCM *theta = (ThetaDCM *) p_theta;
    
    v = exp(x.arr[INDEX_V * x.dim + i] * theta->alpha - theta->tau[i]);
    lnE0 = theta->lnE0;

    //    PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    dq = (1 - exp(f * theta->ln1_E0))*exp(x.arr[INDEX_F * x.dim + i] -
        lnE0 - theta->tau[i] -  x.arr[INDEX_Q * x.dim + i]) - v;

    return dq;
}

__device__ 
MPFLOAT
dcm_lk1(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    MPFLOAT l;
    MPFLOAT q = exp(x.arr[INDEX_Q * x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k1 * ( 1 - q);

    return l;
}


__device__
MPFLOAT
dcm_lk2(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    MPFLOAT l;
    MPFLOAT qv = exp(x.arr[INDEX_Q * x.dim + i] - x.arr[INDEX_V *x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k2 * ( 1 - qv);

    return l;
}

__device__
MPFLOAT
dcm_lk3(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    MPFLOAT l;

    MPFLOAT v = exp(x.arr[INDEX_V * x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k3 * ( 1 - v);

    return l;
}

__device__
void
dcm_upy(dbuff ox, dbuff y, dbuff u, void *theta,
    void *ptheta, dbuff nx)
{
    // Network node
    int j = threadIdx.x%y.dim;

    switch( threadIdx.y )
    {
        case INDEX_LK1 :
            nx.arr[ INDEX_LK1 * nx.dim + j] =
                dcm_lk1(ox, y, u, theta, ptheta, j);
            break;
        case INDEX_LK2:
            nx.arr[ INDEX_LK2 * nx.dim + j] =
                dcm_lk2(ox, y, u, theta, ptheta, j);
            break;
        case INDEX_LK3 :
            nx.arr[ INDEX_LK3 * nx.dim + j] =
                dcm_lk3(ox, y, u, theta, ptheta, j);
            break;
    }

}


