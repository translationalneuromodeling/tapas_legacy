#include "hip/hip_runtime.h"
/* aponteeduardo@gmail.com */
/* copyright (C) 2014 */


#include "mpdcm.hcu"

#define DIM_PTHETA 3
#define DIM_DPTHETA 0

#define DIM_X 5 
#define INDEX_X 0
#define INDEX_F 1
#define INDEX_S 2
#define INDEX_V 3
#define INDEX_Q 4

#define PRELOC_SIZE_X_KR4 4
#define PRELOC_SIZE_X_EULER 2

#define DIM_THETA 7

#define INDEX_V0 0
#define INDEX_E0 1
#define INDEX_K1 2
#define INDEX_K2 3
#define INDEX_K3 4
#define INDEX_ALPHA 5
#define INDEX_GAMMA 6


#define INDEX_LK1 0 
#define INDEX_LK2 1 
#define INDEX_LK3 2 

#define NUM_BLOCKS 16
#define NUM_THREADS 64

#define KRW1 0.16666666666
#define KRW2 0.33333333333
#define KRW3 0.33333333333
#define KRW4 0.16666666666

__device__ void dcm_upx_kr4(dbuff ox, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, dbuff nx);


__device__ double dcm_dx(dbuff x, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, int i)
{
    double dx=0;
    double bt=0;
    int j;
    int k;
    int p;
    int o;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    o = INDEX_X * x.dim;

    // A
    for (j = 0; j < x.dim; j++)
    {
        dx += theta->A[i + x.dim*j] * x.arr[o + j];
    }

    for (j = 0; j < u.dim; j++)
    {
        if (  u.arr[j] == 0  )
            continue;
        // B
        bt = 0;
        k = x.dim*x.dim*j + i;
        for (p = 0; p < x.dim; p++){
            bt += theta->B[k + x.dim*p] * x.arr[o + p];
        }
        // C
        dx += (theta->C[i + x.dim*j] + bt)*u.arr[j];
    }

    return dx;
}

__device__ double dcm_ds(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int i)
{
    double ds;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    ds = x.arr[INDEX_X * x.dim + i] - 
        theta->K[i] * x.arr[INDEX_S * x.dim + i] -
        theta->gamma * (exp(x.arr[INDEX_F * x.dim + i]) - 1);

    return ds;
}

__device__ double dcm_df(dbuff x, dbuff y, dbuff u, void *p_theta, 
    void *p_ptheta, int i)
{
    double df;

    //ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    df = x.arr[INDEX_S * x.dim + i]*exp(-x.arr[INDEX_F * x.dim + i]);

    return df;
}

__device__ double dcm_dv(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int i)
{
    double dv;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    dv = exp(x.arr[INDEX_F * x.dim + i] - x.arr[INDEX_V * x.dim + i] - 
            theta->tau[i]) -
        exp(x.arr[INDEX_V * x.dim + i] * theta->alpha - theta->tau[i]);

    return dv;
}

__device__ double dcm_dq(dbuff x, dbuff y, dbuff u, void *p_theta, 
    void *p_ptheta, int i)
{
    double dq = 0;
    double f = exp(-x.arr[INDEX_F * x.dim + i]);
    double v;
    double lnE0; 
    ThetaDCM *theta = (ThetaDCM *) p_theta;
    
    v = exp(x.arr[INDEX_V * x.dim + i] * theta->alpha - theta->tau[i]);
    lnE0 = theta->lnE0;

    //    PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    //dq = exp(x.arr[INDEX_F * x.dim + i] - x.arr[INDEX_Q * x.dim + i] - lnE0) -
    //    (exp(x.arr[INDEX_F * x.dim + i] + theta->ln1_E0*f - 
    //        x.arr[INDEX_Q * x.dim + i] - lnE0))  -  v;

    dq = (1 - exp(f * theta->ln1_E0))*exp(x.arr[INDEX_F * x.dim + i] -
        lnE0 - theta->tau[i] -  x.arr[INDEX_Q * x.dim + i]) - v;

    return dq;
}

__device__ double dcm_lk1(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    double l;
    double q = exp(x.arr[INDEX_Q * x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k1 * ( 1 - q);

    return l;
}


__device__ double dcm_lk2(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    double l;
    double qv = exp(x.arr[INDEX_Q * x.dim + i] - x.arr[INDEX_V *x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k2 * ( 1 - qv);

    return l;
}

__device__ double dcm_lk3(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    double l;

    double v = exp(x.arr[INDEX_V * x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k3 * ( 1 - v);

    return l;
}

__device__ void dcm_upy(dbuff ox, dbuff y, dbuff u, void *theta,
    void *ptheta, dbuff nx)
{
    // Network node
    int j = threadIdx.x%y.dim;

    switch( threadIdx.y )
    {
        case INDEX_LK1 :
            nx.arr[ INDEX_LK1 * nx.dim + j] =
                dcm_lk1(ox, y, u, theta, ptheta, j);
            break;
        case INDEX_LK2:
            nx.arr[ INDEX_LK2 * nx.dim + j] =
                dcm_lk2(ox, y, u, theta, ptheta, j);
            break;
        case INDEX_LK3 :
            nx.arr[ INDEX_LK3 * nx.dim + j] =
                dcm_lk3(ox, y, u, theta, ptheta, j);
            break;
    }

}

__device__ void dcm_upx_euler(dbuff ox, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, dbuff nx)
{

    //ThetaDCM *theta = (ThetaDCM *) p_theta;
    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    int j = threadIdx.x%y.dim;
    int s;

    // Make the values to be closer in range
 
    if ( isnan( *u.arr ) ){
        if ( threadIdx.y == 0 )
        {
            nx.arr[ INDEX_X * ox.dim + j] = NAN;
            nx.arr[ INDEX_F * ox.dim + j] = NAN;
            nx.arr[ INDEX_S * ox.dim + j] = NAN;
            nx.arr[ INDEX_V * ox.dim + j] = NAN;
            nx.arr[ INDEX_Q * ox.dim + j] = NAN;
        }
    }

    switch ( threadIdx.y )
    {
        case INDEX_X:
            s = INDEX_X * ox.dim + j;
            nx.arr[s] = ox.arr[s] + 
                ptheta->de * dcm_dx(ox, y, u, p_theta, p_ptheta, j);
            break;
        case INDEX_F:
            s = INDEX_F * ox.dim + j;
            nx.arr[s] = ox.arr[s] + 
                ptheta->de * dcm_df(ox, y, u, p_theta, p_ptheta, j);
            break;
        case INDEX_S:
            s = INDEX_S * ox.dim + j;
            nx.arr[s] = ox.arr[s] + 
                ptheta->de * dcm_ds(ox, y, u, p_theta, p_ptheta, j);
            break;
        case INDEX_V:
            s = INDEX_V * ox.dim + j;
            nx.arr[s] = ox.arr[s] + 
                ptheta->de * dcm_dv(ox, y, u, p_theta, p_ptheta, j);
            break;
        case INDEX_Q:
            s = INDEX_Q * ox.dim + j;
            nx.arr[s] = ox.arr[s] + 
                ptheta->de * dcm_dq(ox, y, u, p_theta, p_ptheta, j); 
            break;
    }

}

__device__ void dcm_int_euler(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int dp)
{
    int i;
    int j = threadIdx.x%y.dim;
    double *t;
    // Number of integration steps done between each data point
    int ss, dy;
    // Point where threads are not synchronized to anything
    int maxx = y.dim * (blockDim.x/y.dim);

    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;
    dbuff ox;
    dbuff nx;

    dbuff ty;
    dbuff tu;


    ox.dim = y.dim;
    nx.dim = y.dim;

    ox.arr = x.arr; 
    nx.arr = ox.arr + nx.dim * DIM_X;

    if ( threadIdx.x < maxx )
        memset(x.arr, 0, nx.dim * DIM_X * sizeof(double));

    __syncthreads();
    ty.dim = y.dim;
    tu.dim = u.dim;

    // How many samples are gonna be taken
    ss = ceil(1.0/ptheta->dt);
    dy = ceil(1.0/(ptheta->dt * ptheta->dyu));

    ty.arr = y.arr; 
    tu.arr = u.arr;

    for (i=0; i < dp*ss; i++)
    {
        if ( threadIdx.x < maxx )
            dcm_upx_euler(ox, ty, tu, p_theta, p_ptheta, nx);
        __syncthreads();
        // Only sample every 1/ptheta->dt times
        if ( i%ss == 0 )
        {
            if ( i%dy == (dy-2) ) 
           {
                if ( threadIdx.x < maxx )
                    dcm_upy(nx, ty, tu, p_theta, p_ptheta, ox);           
                __syncthreads();
                if ( threadIdx.x < maxx && threadIdx.y == 0 )
                    ty.arr[j] = ox.arr[INDEX_LK1 * ox.dim + j] +
                        ox.arr[ INDEX_LK2 * ox.dim + j] +
                        ox.arr[ INDEX_LK3 * ox.dim + j];
                __syncthreads();

                ty.arr += y.dim; 
            }
            if ( i > 0 )
                tu.arr += u.dim;
        }
        // Swap the pointers
        t = ox.arr;
        ox.arr = nx.arr;
        nx.arr = t;
    }
}

__device__ void dcm_int_kr4(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int dp)
{
    int i;
    int j = threadIdx.x%y.dim;
    double *t;
    // Number of integration steps done between each data point
    int ss, dy;
    // Point where threads are not synchronized to anything
    int maxx = y.dim * (blockDim.x/y.dim);

    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;
    dbuff ox;
    dbuff nx;

    dbuff ty;
    dbuff tu;


    ox.dim = y.dim;
    nx.dim = y.dim;

    ox.arr = x.arr; 
    nx.arr = ox.arr + nx.dim * DIM_X;

    if ( threadIdx.x < maxx )
        memset(x.arr, 0, nx.dim * DIM_X * sizeof(double));

    __syncthreads();
    ty.dim = y.dim;
    tu.dim = u.dim;

    // How many samples are gonna be taken
    ss = ceil(1.0/ptheta->dt);
    dy = ceil(1.0/(ptheta->dt * ptheta->dyu));

    ty.arr = y.arr; 
    tu.arr = u.arr;

    for (i=0; i < dp*ss; i++)
    {
        //if ( threadIdx.x < maxx )
            dcm_upx_kr4(ox, ty, tu, p_theta, p_ptheta, nx);
        __syncthreads();
        // Only sample every 1/ptheta->dt times
        if ( i%ss == 0 )
        {
            if ( i%dy == (dy-2) ) 
            {
                if ( threadIdx.x < maxx )
                    dcm_upy(nx, ty, tu, p_theta, p_ptheta, ox);           
                __syncthreads();
                if ( threadIdx.x < maxx && threadIdx.y == 0 )
                    ty.arr[j] = ox.arr[INDEX_LK1 * ox.dim + j] +
                        ox.arr[ INDEX_LK2 * ox.dim + j] +
                        ox.arr[ INDEX_LK3 * ox.dim + j];
                if ( i > 0 )
                    ty.arr += y.dim;
               __syncthreads(); 
            }
            if ( i > 0 )
                tu.arr += u.dim;
        }
        // Swap the pointers
        t = ox.arr;
        ox.arr = nx.arr;
        nx.arr = t;
    }
}

// ==========================================================================
// Kernel code
// ==========================================================================

__global__ void kdcm_fmri_euler(double *x, double *y, double *u, 
    void *p_theta, double *d_theta, void *p_ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb)
{
    /* 
    mem -- Prealocate shared memory. It depends on the slots that the 
        integrator needs; two for euler and 4 for Kutta-Ruge.
    fupx -- Function used to integrate the update the system. 
    */

    int i;
    dbuff tx, ty, tu;
    __shared__ double sx[NUM_THREADS * DIM_X * PRELOC_SIZE_X_EULER];

    // Assign pointers to theta


    ThetaDCM *theta = (ThetaDCM *) p_theta;
    ThetaDCM *ltheta;

    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;
    __shared__ PThetaDCM lptheta[1];

    lptheta->dt = ptheta->dt;
    lptheta->dyu = ptheta->dyu;
    lptheta->de = ptheta->de;
    lptheta->mode = ptheta->mode;

    tu.dim = nu;
    tx.dim = nx; 
    ty.dim = nx;

    i = threadIdx.x/nx + (blockDim.x / nx) * blockIdx.x;
    // Iterate in chuncks
    while ( i < nb * nt )
    {
        double *o;

        tu.arr = u + (i/nb) * nu * dp;
        // Get the new address

        ltheta = theta + i;

        o = d_theta + i * (
            nx * nx + // A
            nx * nx * nu + // B 
            nx * nu + // C
            nx + // Kappa (K)
            nx); // tau
        
        ltheta->A = o;
        o += nx * nx;

        ltheta->B = o;
        o += nx * nx * nu;

        ltheta->C = o; 
        o+= nx * nu;

        ltheta->K = o;
        o += nx;

        ltheta->tau = o; 

        tx.arr = sx + PRELOC_SIZE_X_EULER * DIM_X * nx * (threadIdx.x/nx);

        ty.arr = y + i * nx * ny;
        dcm_int_euler(tx, ty, tu, (void *) ltheta, (void *) lptheta, dp);
        i += gridDim.x * (blockDim.x / nx );        
    }
}

__global__ void kdcm_fmri_kr4(double *x, double *y, double *u, 
    void *p_theta, double *d_theta, void *p_ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb)
{
    /* 
    mem -- Prealocate shared memory. It depends on the slots that the 
        integrator needs; two for euler and 4 for Kutta-Ruge.
    fupx -- Function used to integrate the update the system. 
    */

    int i;
    dbuff tx, ty, tu;
    __shared__ double sx[NUM_THREADS * DIM_X * PRELOC_SIZE_X_KR4];

    // Assign pointers to theta


    ThetaDCM *theta = (ThetaDCM *) p_theta;
    ThetaDCM *ltheta;

    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;
    __shared__ PThetaDCM lptheta[1];

    lptheta->dt = ptheta->dt;
    lptheta->dyu = ptheta->dyu;
    lptheta->de = ptheta->de;
    lptheta->mode = ptheta->mode;

    tu.dim = nu;
    tx.dim = nx; 
    ty.dim = nx;

    i = threadIdx.x/nx + (blockDim.x / nx) * blockIdx.x;
    // Iterate in chuncks
    while ( i < nb * nt )
    {
        double *o;

        tu.arr = u + (i/nb) * nu * dp;
        // Get the new address

        ltheta = theta + i;

        o = d_theta + i * (
            nx * nx + // A
            nx * nx * nu + // B 
            nx * nu + // C
            nx + // Kappa (K)
            nx); // tau
        
        ltheta->A = o;
        o += nx * nx;

        ltheta->B = o;
        o += nx * nx * nu;

        ltheta->C = o; 
        o+= nx * nu;

        ltheta->K = o;
        o += nx;

        ltheta->tau = o; 

        tx.arr = sx + PRELOC_SIZE_X_KR4 * DIM_X * nx * (threadIdx.x/nx);

        ty.arr = y + i * nx * ny;
        dcm_int_kr4(tx, ty, tu, (void *) ltheta, (void *) lptheta, dp);
        i += gridDim.x * (blockDim.x / nx );        
    }
}


// ===========================================================================
// Kernel (l)auncher
// ===========================================================================

__host__ void ldcm_fmri_euler(double *x, double *y, double *u, 
    void *theta, double *d_theta, void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb )
{

    dim3 gthreads(NUM_THREADS, DIM_X);
    dim3 gblocks(NUM_BLOCKS/PRELOC_SIZE_X_EULER, 1);

    kdcm_fmri_euler<<<gblocks, gthreads>>>(x, y, u, 
        theta, d_theta, ptheta, d_ptheta, 
        nx, ny, nu, dp, nt, nb ); 
}


__host__ void ldcm_fmri_kr4(double *x, double *y, double *u, 
    void *theta, double *d_theta, void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb )
{

    dim3 gthreads(NUM_THREADS, DIM_X);
    dim3 gblocks(NUM_BLOCKS/PRELOC_SIZE_X_KR4, 1);

    kdcm_fmri_kr4<<<gblocks, gthreads>>>(x, y, u, 
        theta, d_theta, ptheta, d_ptheta, 
        nx, ny, nu, dp, nt, nb ); 
}


// ===========================================================================
// Allocate memory
// ===========================================================================

// Device alloc memory theta
__host__ void dam_theta(
    void **theta, double **d_theta,
    void **pd_theta, double **dd_theta,
    int nx, int ny, int nu, int dp, int nt, int nb)
{

    int tp;

    // Allocate memory for the structures

    HANDLE_ERROR( hipMalloc( pd_theta, nt * nb * sizeof(ThetaDCM)));
    HANDLE_ERROR( hipMemcpy( *pd_theta, *theta, nt * nb * sizeof(ThetaDCM),
        hipMemcpyHostToDevice ) );

    // Allocate memory for the matrices. It is assumed that the parameters
    // are in a block of contiguous memory with the for A, Bs, C, x0, epsilon,
    // Kappa, tau

    tp = nt * nb * 
        (nx * nx +      // A:
        nx * nx * nu +  // B's
        nx * nu +       // C
        nx + // kappa
        nx); // tau 

    HANDLE_ERROR( hipMalloc( dd_theta, tp * sizeof(double) ) );
    HANDLE_ERROR( hipMemcpy( (void *) *dd_theta, (void *) *d_theta, 
        tp * sizeof(double), hipMemcpyHostToDevice ) );

}

// Device alloc memory ptheta
__host__ void dam_ptheta(
    void **ptheta, double **d_ptheta, 
    void **pd_ptheta, double **dd_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb)
{

    HANDLE_ERROR( hipMalloc( pd_ptheta, sizeof(PThetaDCM)));
    HANDLE_ERROR( hipMemcpy( *pd_ptheta, *ptheta, sizeof(PThetaDCM),
        hipMemcpyHostToDevice ) );
 
}


// Host code
extern "C"
int mpdcm_fmri( double *x, double *y, double *u,
    void *theta, double *d_theta,
    void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb,
    klauncher launcher)
{

    double *d_x, *d_y, *d_u;
    void *pd_theta, *pd_ptheta;
    double *dd_theta, *dd_ptheta;

    // x

    d_x = 0;

    // y

    HANDLE_ERROR( hipMalloc( (void **) &d_y,
        nx * ny * nt * nb * sizeof(double) ) );

    // u

    HANDLE_ERROR( hipMalloc( (void**) &d_u,
        nt * nu * dp *  sizeof(double) ) );
    HANDLE_ERROR( hipMemcpy( d_u, u, nt * nu * dp * sizeof(double),
        hipMemcpyHostToDevice ) );

    // Theta 
    dam_theta(
        &theta, &d_theta,
        &pd_theta, &dd_theta,
        nx, ny, nu, dp, nt, nb);
    
    // PThetaDCM

    dam_ptheta(
        &ptheta, &d_ptheta,
        &pd_ptheta, &dd_ptheta,
        nx, ny, nu, dp, nt, nb); 

    // Launch the kernel
    (*launcher)(
        d_x, d_y, d_u, 
        pd_theta, dd_theta, 
        pd_ptheta, dd_ptheta,
        nx, ny, nu, dp, nt, nb );

    // Get y back

    HANDLE_ERROR( hipMemcpy(y, d_y,
        nx * ny * nt * nb * sizeof(double),
        hipMemcpyDeviceToHost) );


    // free the memory allocated on the GPU
    //HANDLE_ERROR( hipFree( d_x ) );
    HANDLE_ERROR( hipFree( d_y ) );
    HANDLE_ERROR( hipFree( d_u ) );

    HANDLE_ERROR( hipFree( pd_theta ) );
    HANDLE_ERROR( hipFree( dd_theta ) );

    if ( DIM_PTHETA ) HANDLE_ERROR( hipFree( pd_ptheta ) );
    if ( DIM_DPTHETA ) HANDLE_ERROR( hipFree( dd_ptheta ) );
    
    return 0; 
}

extern "C"
int mpdcm_fmri_euler( double *x, double *y, double *u,
    void *theta, double *d_theta,
    void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb)
{
   int r = mpdcm_fmri(x, y, u,
        theta, d_theta,
        ptheta, d_ptheta, 
        nx, ny, nu, dp, nt, nb,
        &ldcm_fmri_euler);
    
    return r;
};

extern "C"
int mpdcm_fmri_kr4( double *x, double *y, double *u,
    void *theta, double *d_theta,
    void *ptheta, double *d_ptheta, 
    int nx, int ny, int nu, int dp, int nt, int nb)
{
    int r = mpdcm_fmri(x, y, u,
        theta, d_theta,
        ptheta, d_ptheta, 
        nx, ny, nu, dp, nt, nb,
        &ldcm_fmri_kr4);

    return r;
};



// =======================================================================


__device__ void dcm_upx_kr4(dbuff ox, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, dbuff nx)
{

    //ThetaDCM *theta = (ThetaDCM *) p_theta;
    PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    int j = threadIdx.x%y.dim;
    int s;
    dbuff k1, k2;
    int maxx = y.dim * (blockDim.x/y.dim);


    k1.arr = (ox.arr < nx.arr) ? ox.arr : nx.arr;
    k2.arr = k1.arr;
    k1.arr += ox.dim * DIM_X * 2;  
    k2.arr += ox.dim * DIM_X * 3;

    k1.dim = ox.dim;
    k2.dim = ox.dim;
    // Make the values to be closer in range
 
    if ( isnan( *u.arr ) ){
        if ( threadIdx.y == 0 )
        {
            nx.arr[ INDEX_X * ox.dim + j] = NAN;
            nx.arr[ INDEX_F * ox.dim + j] = NAN;
            nx.arr[ INDEX_S * ox.dim + j] = NAN;
            nx.arr[ INDEX_V * ox.dim + j] = NAN;
            nx.arr[ INDEX_Q * ox.dim + j] = NAN;
        }
    }

    if ( threadIdx.x < maxx )
    {
    switch ( threadIdx.y )
    {
        case INDEX_X:
            s = INDEX_X * ox.dim + j;
            k1.arr[s] = dcm_dx(ox, y, u, p_theta, p_ptheta, j);
            nx.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s] * KRW1;
            k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
            break;
        case INDEX_F:
            s = INDEX_F * ox.dim + j;
            k1.arr[s] = dcm_df(ox, y, u, p_theta, p_ptheta, j);
            nx.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s] * KRW1;
            k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
            break;
        case INDEX_S:
            s = INDEX_S * ox.dim + j;
            k1.arr[s] = dcm_ds(ox, y, u, p_theta, p_ptheta, j);
            nx.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s] * KRW1;
            k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
            break;
        case INDEX_V:
            s = INDEX_V * ox.dim + j;
            k1.arr[s] = dcm_dv(ox, y, u, p_theta, p_ptheta, j);
            nx.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s] * KRW1;
            k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
            break;
        case INDEX_Q:
            s = INDEX_Q * ox.dim + j;
            k1.arr[s] = dcm_dq(ox, y, u, p_theta, p_ptheta, j); 
            nx.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s] * KRW1;
            k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
            break;
    }
    }
   __syncthreads();

    if ( threadIdx.x < maxx )
    {
    switch ( threadIdx.y )
    {
        case INDEX_X:
            k1.arr[s] = dcm_dx(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s]*KRW2;
            break;
        case INDEX_F:
            k1.arr[s] = dcm_df(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s]*KRW2;
            break;
        case INDEX_S:
            k1.arr[s] = dcm_ds(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s]*KRW2;
            break;
        case INDEX_V:
            k1.arr[s] = dcm_dv(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s]*KRW2;
            break;
        case INDEX_Q:
            k1.arr[s] = dcm_dq(k2, y, u, p_theta, p_ptheta, j); 
            nx.arr[s] += ptheta->de * k1.arr[s]*KRW2;
            break;
    }
    }
    __syncthreads();
    if ( threadIdx.x < maxx )
    {
    switch ( threadIdx.y )
    {
        case INDEX_X:
            k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
            break;
        case INDEX_F:
            k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
            break;
        case INDEX_S:
            k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
            break;
        case INDEX_V:
            k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
            break;
        case INDEX_Q:
            k2.arr[s] = ox.arr[s] + ptheta->de * 0.5 * k1.arr[s]; 
            break;
    }
    }
    __syncthreads();
    if ( threadIdx.x < maxx )
    {
    switch ( threadIdx.y )
    {
        case INDEX_X:
            k1.arr[s] = dcm_dx(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s]*KRW3;
            break;
        case INDEX_F:
            k1.arr[s] = dcm_df(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s]*KRW3;
            break;
        case INDEX_S:
            k1.arr[s] = dcm_ds(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s]*KRW3;
            break;
        case INDEX_V:
            k1.arr[s] = dcm_dv(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s]*KRW3;
            break;
        case INDEX_Q:
            k1.arr[s] = dcm_dq(k2, y, u, p_theta, p_ptheta, j); 
            nx.arr[s] += ptheta->de * k1.arr[s]*KRW3;
            break;
    }
    }
    __syncthreads();
    if ( threadIdx.x < maxx )
    {
    switch ( threadIdx.y )
    {
        case INDEX_X:
            k2.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s]; 
            break;
        case INDEX_F:
            k2.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s]; 
            break;
        case INDEX_S:
            k2.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s]; 
            break;
        case INDEX_V:
            k2.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s]; 
            break;
        case INDEX_Q:
            k2.arr[s] = ox.arr[s] + ptheta->de * k1.arr[s]; 
            break;
    }
    }
    __syncthreads();
    if ( threadIdx.x < maxx )
    {
    switch ( threadIdx.y )
    {
        case INDEX_X:
            k1.arr[s] = dcm_dx(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s] * KRW4;
            break;
        case INDEX_F:
            k1.arr[s] = dcm_df(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s] * KRW4;
            break;
        case INDEX_S:
            k1.arr[s] = dcm_ds(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s] * KRW4;
            break;
        case INDEX_V:
            k1.arr[s] = dcm_dv(k2, y, u, p_theta, p_ptheta, j);
            nx.arr[s] += ptheta->de * k1.arr[s] * KRW4;
            break;
        case INDEX_Q:
            k1.arr[s] = dcm_dq(k2, y, u, p_theta, p_ptheta, j); 
            nx.arr[s] += ptheta->de * k1.arr[s] * KRW4;
            break;
    }
    //__syncthreads();
    }
}
